#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include ""


float* a, * b, * c; //host variables

//Incluindo modificação na branch

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    
    //Calcula o indice da linha do elemento P e M
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
 
    //Calcula o indice da coluna do elemento P e N
    int Col = blockIdx.x*blockDim.x + threadIdx.x;

    if ((Row < Width) && (Col < Width)) {
        float Pvalue = 0;

        //Cada thread calcula um elemento do bloco da submatriz
        for (int k = 0; k < Width; k++) {
            Pvalue += M[Row * Width + k] * N[k * Width + Col];
        }

        P[Row * Width + Col] = Pvalue;
    }
}

int main()
{
    hipDeviceReset();

    float* d_a, * d_b, * d_c; //device variables

    int n = 16;
    int size = n * n * sizeof(float);
    dim3 dimGrid((n-1)/16 + 1, (n - 1) / 16 + 1, 1);
    dim3 dimBlock(16, 16, 1);


    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);
   
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    for (int i = 0; i < n*n; i++)
        a[i] = 2, b[i] = 2;

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    
    MatrixMulKernel << <dimGrid,dimBlock >> > (d_a, d_b, d_c, n);
    hipDeviceSynchronize;

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    //   for (int i = 0; i < n; i++)
        printf("%f ", c[2]);

    hipFree(d_a), hipFree(d_b), hipFree(d_c);

    return 0;
}
